#include "hip/hip_runtime.h"
﻿#include "SandParticle.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include "Affine.h"
#include <atomic>
#include <vector>
const int width = 100;
const int height = 10;
const int gridWidth = 800;
const int gridHeight = 800;
const dim3 threadsPerBlock(16, 16);
const int blockSize = 256;
__constant__ float gap = 0.01f;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define cudaCheck(x) { hipError_t err = x; if (err != hipSuccess) { printf("Cuda error: %d in %s at %s:%d\n", err, #x, __FILE__, __LINE__); assert(0); } }

__global__ void SetGrid(ParticleGrid* grid)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
	float xPos = -5 + ((index % gridWidth) * gap);
	float yPos = ((index / gridHeight) * gap);

	grid[index].gridPos.x = xPos;
	grid[index].gridPos.y = yPos;
	grid[index].index = index;
	grid[index].status = Empty;
}

__global__ void SetSpawner(ParticleGrid* grid, SpawnerPos* spawnerPos, int startIndex, int width, int i)
{
	int gridIndex = startIndex + threadIdx.x + (i * gridWidth);
	ParticleGrid& currGrid = grid[gridIndex];
	int iIndex = i * width;

	//currGrid.status = FilledWithLand;
	spawnerPos[iIndex + threadIdx.x].pos = currGrid.gridPos;
	spawnerPos[iIndex + threadIdx.x].currGridIndex = currGrid.index;
}

__global__ void SetLand(ParticleGrid* grid, Land* land, int landStartIndex, int landWidth, int i)
{

	int index = landStartIndex + threadIdx.x;
	int ogLandStartIndex = landStartIndex;
	ParticleGrid& currGrid = grid[index];
	int iIndex = i * landWidth;

	currGrid.status = FilledWithLand;
	land[iIndex + threadIdx.x].landPos = currGrid.gridPos;

}

__global__ void CheckGridPos(ParticleSand* particles, ParticleGrid* grids)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	const int startIndex = gridWidth * 500 + 400;

	const int widthVal = index % width;
	const int heightVal = index / width;
	const int heightInGrid = heightVal * gridWidth;
	

	particles[index].currGrid = &grids[startIndex + heightInGrid + widthVal];
	particles[index].pos = grids[startIndex + heightInGrid + widthVal].gridPos;

}

__global__ void SpawnerMove(ParticleGrid* grids, SpawnerPos* spawnerPos, int dir)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	const int currGridIndex = spawnerPos[index].currGridIndex;

	//printf("currGrid : %d", currGridIndex);

	const int destGridIndex = currGridIndex + dir;

	SpawnerPos& spawner = spawnerPos[index];

	spawner.currGridIndex = destGridIndex;
	spawner.pos = grids[destGridIndex].gridPos;
}

__global__ void Spawn(ParticleSand* particle, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	int index = lastIndex + threadIdx.x;

	ParticleSand& part = particle[index];
	SpawnerPos& spawner = spawners[threadIdx.x];
	ParticleGrid& gridInfo = grid[spawner.currGridIndex];

	part.pos = spawner.pos;
	part.currGrid = &grid[spawner.currGridIndex];
	gridInfo.status = FilledWithSand;
}

__global__ void SpawnLand(Land* lands, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	int index = lastIndex + threadIdx.x + (blockIdx.x * blockDim.x);

	Land& land = lands[index];
	SpawnerPos& spawner = spawners[threadIdx.x];
	ParticleGrid& gridInfo = grid[spawner.currGridIndex];

	land.landPos = spawner.pos;
	land.currGridIndex = spawner.currGridIndex;
	gridInfo.status = FilledWithLand;
}




__global__ void MoveDown(ParticleSand* particles, ParticleGrid* grids, int particleNum)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= particleNum)
	{
		return;
	}

	ParticleSand& particle = particles[index];
	ParticleGrid& currGrid = *particle.currGrid;

	const int gridIndex = currGrid.index;
	int gridDownIndex = gridIndex - gridWidth;
	int gridLeftDownIndex = gridIndex - (gridWidth - 1);
	int gridRightDownIndex = gridIndex - (gridWidth + 1);

	if (gridIndex < gridWidth)
	{
		gridDownIndex = gridIndex;
		gridLeftDownIndex = gridIndex;
		gridRightDownIndex = gridIndex;
	}
	if (gridIndex % gridWidth == 0)
	{
		gridLeftDownIndex = gridIndex;
	}
	if (gridIndex % gridWidth == gridWidth - 1)
	{
		gridRightDownIndex = gridIndex;
	}

	if (gridDownIndex <= 0 || gridDownIndex >= 256 * 2000 ||
		gridLeftDownIndex <= 0 || gridLeftDownIndex >= 256 * 2000 ||
		gridRightDownIndex <= 0 || gridRightDownIndex >= 256 * 2000)
	{
		return;
	}

	ParticleGrid& downGrid = grids[gridDownIndex];
	ParticleGrid& leftDownGrid = grids[gridLeftDownIndex];
	ParticleGrid& rightDownGrid = grids[gridRightDownIndex];

	if (downGrid.status == Empty)
	{
		particle.pos = downGrid.gridPos;
		particle.currGrid = &downGrid;

		currGrid.status = Empty;
		downGrid.status = FilledWithSand;
	}
	else if (leftDownGrid.status == Empty)
	{
		particle.pos = leftDownGrid.gridPos;
		particle.currGrid = &leftDownGrid;

		currGrid.status = Empty;
		leftDownGrid.status = FilledWithSand;
	}
	else if (rightDownGrid.status == Empty)
	{
		particle.pos = rightDownGrid.gridPos;
		particle.currGrid = &rightDownGrid;

		currGrid.status = Empty;
		rightDownGrid.status = FilledWithSand;
	}
}
void SandUpdate(int particleNum, int gridNum, ParticleSand* particle, ParticleGrid* grid)
{
	int gridCount = particleNum / blockSize;
	MoveDown << <gridCount, blockSize>> > (particle, grid, particleNum);
	gpuErrchk(hipPeekAtLastError());
}

void AddSandsInSpawnerPos(ParticleSand* particle, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	Spawn << <1, blockSize >> > (particle, grid, spawners, lastIndex);
}
void AddLandsInSpawnerPos(Land* lands, ParticleGrid* grid, SpawnerPos* spawners, int lastIndex)
{
	SpawnLand << <1, blockSize >> > (lands, grid, spawners, lastIndex);
}

void MoveSpawner(ParticleGrid* grid, SpawnerPos* spawners, int dir, int spawnerCount)
{
	SpawnerMove << <1, spawnerCount >> > (grid, spawners, dir);
}



void SetLands(std::vector<int>& landStartRandomIndices, ParticleSand* particle, ParticleGrid* grid, Land* land)
{
	size_t vecSize = landStartRandomIndices.size();
	int landCount = 1;
	int landWidth = 50;
	int landStartIndex = gridWidth * 100 + 200;

	for (size_t i = 0; i < vecSize; ++i)
	{
		SetLand << <landCount, landWidth >> > (grid, land, landStartRandomIndices[i], landWidth, i);
	}

}
__global__ void LoadLand(Land* lands, ParticleGrid* grid, int* indices, int lastIndex)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index >= lastIndex)
	{
		printf("over");
		return;
	}
	int gridIndex = indices[index];
	lands[index].landPos = grid[gridIndex].gridPos;
	lands[index].currGridIndex = grid[gridIndex].index;
	grid[gridIndex].status = FilledWithLand;
}

void LoadLands(int* landGridIndices, ParticleGrid* grid, Land* land, int landsNum)
{
	int landBlockCount = landsNum / blockSize;

	if (landBlockCount < 0)
		landBlockCount = 1;

	LoadLand << <landBlockCount, blockSize>> > (land, grid, landGridIndices, landsNum);
	gpuErrchk(hipPeekAtLastError());
}

void Init(int particleNum, int gridNum, int spawnerNum, ParticleSand* particle, ParticleGrid* grid, Land* land, SpawnerPos* spawners)
{
	int particleGridCount = particleNum / blockSize;
	int gridCount = gridNum / blockSize;
	
	int spawnerStartIndex = gridWidth * 300 + 500;

	SetGrid << <gridCount, blockSize >> > (grid);
	CheckGridPos << <particleGridCount, blockSize >> > (particle, grid);

	int spawnerWidth = sqrt(spawnerNum);
	int spwanerCount = 1;

	for (int i = 0; i < spawnerWidth; ++i)
	{
		SetSpawner << <spwanerCount, spawnerWidth >> > (grid, spawners, spawnerStartIndex, spawnerWidth, i);
	}
	gpuErrchk(hipPeekAtLastError());
}