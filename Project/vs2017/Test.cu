#include "hip/hip_runtime.h"
﻿#include "SandParticle.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include "Affine.h"
#include <atomic>
const int width = 50;
const int height = 10;
const int gridWidth = 100;
const int gridHeight = 100;
const dim3 threadsPerBlock(16, 16);
const int blockSize = 256;
__constant__ float gap = 0.05f;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define cudaCheck(x) { hipError_t err = x; if (err != hipSuccess) { printf("Cuda error: %d in %s at %s:%d\n", err, #x, __FILE__, __LINE__); assert(0); } }

__global__ void SetGrid(ParticleGrid* grid)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
	float xPos = ((index % gridWidth) * gap);
	float yPos = ((index / gridHeight) * gap);

	grid[index].gridPos.x = xPos;
	grid[index].gridPos.y = yPos;
	grid[index].index = index;
	grid[index].status = Empty;
}

__global__ void SetLand(ParticleGrid* grid, Land* land)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	ParticleGrid& currGrid = grid[index];
	int landStartIndex = gridWidth * 5 + 10;
	int landWidth = 50;
	int ogLandStartIndex = landStartIndex;
	if (index > landStartIndex && index < landStartIndex + landWidth)
	{
		currGrid.status = FilledWithLand;
		land[index - ogLandStartIndex].landPos = currGrid.gridPos;
	}

	landStartIndex += gridWidth;
	ogLandStartIndex += landWidth;
	if (index > landStartIndex && index < landStartIndex + landWidth)
	{
		currGrid.status = FilledWithLand;
		land[index - ogLandStartIndex].landPos = currGrid.gridPos;
	}

	landStartIndex += gridWidth;
	ogLandStartIndex += landWidth;

	if (index > landStartIndex && index < landStartIndex + landWidth)
	{
		currGrid.status = FilledWithLand;
		land[index - ogLandStartIndex].landPos = currGrid.gridPos;
	}

	landStartIndex += gridWidth;
	ogLandStartIndex += landWidth;

	if (index > landStartIndex && index < landStartIndex + landWidth)
	{
		currGrid.status = FilledWithLand;
		land[index - ogLandStartIndex].landPos = currGrid.gridPos;
	}

	/*if (isInside)
	{
		
	}*/
}

__global__ void CheckGridPos(ParticleSand* particles, ParticleGrid* grids)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
	printf("index : %d\n", index);

	const int startIndex = gridWidth * 10 + 5;

	const int widthVal = index % width;
	const int heightVal = index / width;
	const int heightInGrid = heightVal * gridWidth;
	

	particles[index].currGrid = &grids[startIndex + heightInGrid + widthVal];
	particles[index].pos = grids[startIndex + heightInGrid + widthVal].gridPos;

}

__global__ void MoveDown(ParticleSand* particles, ParticleGrid* grids, int particleNum)
{
	const int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= particleNum)
	{
		return;
	}

	ParticleSand& particle = particles[index];
	ParticleGrid& currGrid = *particle.currGrid;

	const int gridIndex = currGrid.index;

	const int downGridIndex = gridIndex - gridWidth;
	const int downDoubleGridIndex = downGridIndex - gridWidth;
	ParticleGrid& destGrid = grids[downGridIndex];
	ParticleGrid& destDoubleGrid = grids[downDoubleGridIndex];
	
	
	if(destGrid.status == Empty)
	{
		particle.pos = destGrid.gridPos;
		particle.currGrid = &destGrid;

		currGrid.status = Empty;
		destGrid.status = FilledWithSand;
	}
	else if(destGrid.status == FilledWithSand && destDoubleGrid.status == FilledWithLand)
	{
		bool isIndexOdd = index % 2;
		int destGridIndex;
		
		if (isIndexOdd)
		{
			destGridIndex = gridIndex - 1;

			if (gridIndex % gridWidth == 0)
			{
				return;
			}
		}
		else
		{
			destGridIndex = gridIndex + 1;

			if (gridIndex % gridWidth == gridWidth - 1)
			{
				return;
			}
		}
		ParticleGrid& newDestGrid = grids[destGridIndex];
		particle.pos = newDestGrid.gridPos;
		particle.currGrid = &newDestGrid;
		currGrid.status = Empty;
		newDestGrid.status = FilledWithSand;
	}
}
void SandUpdate(int particleNum, int gridNum, ParticleSand* particle, ParticleGrid* grid)
{
	int gridCount = particleNum / blockSize;
	MoveDown << <gridCount, blockSize>> > (particle, grid, particleNum);
	gpuErrchk(hipPeekAtLastError());
}

void Init(int particleNum, int gridNum, ParticleSand* particle, ParticleGrid* grid, Land* land)
{
	int particleGridCount = particleNum / blockSize;
	int gridCount = gridNum / blockSize;

	SetGrid << <gridCount, blockSize >> > (grid);
	SetLand << <gridCount, blockSize >> > (grid, land);
	CheckGridPos << <particleGridCount, blockSize >> > (particle, grid);
}